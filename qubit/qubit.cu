#include "hip/hip_runtime.h"
/*
 * qubit algorithm
 *
 */
extern "C" {
#include "sph/sph_luffa.h"
#include "sph/sph_bmw.h"
#include "sph/sph_shavite.h"
#include "sph/sph_simd.h"
#include "sph/sph_echo.h"
#include "sph/sph_skein.h"
#include "sph/sph_jh.h"
#include "sph/sph_keccak.h"
}

#include "miner.h"

#include "cuda_helper.h"
#include "x11/cuda_x11.h"

static uint32_t *d_hash[MAX_GPUS];

extern void skein512_cpu_setBlock_80(void *pdata);
extern void skein512_cpu_hash_80(int thr_id, uint32_t threads, uint32_t startNonce, uint32_t *d_hash, int swap);
extern void x11_luffa512_cpu_init(int thr_id, uint32_t threads);
//extern void qubit_luffa512_cpu_setBlock_80(void *pdata);
extern void x11_luffa512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash, int order);

extern "C" void qubithash(void *state, const void *input)
{
	uint8_t _ALIGN(128) hash[64];

	// luffa1-cubehash2-shavite3-simd4-echo5

	sph_skein512_context ctx_skein;
	sph_jh512_context ctx_jh;
	sph_keccak512_context ctx_keccak;
	sph_bmw512_context ctx_bmw;
	sph_luffa512_context ctx_luffa;
	sph_echo512_context ctx_echo;
	sph_simd512_context ctx_simd;
	sph_shavite512_context ctx_shavite;

	sph_skein512_init(&ctx_skein);
	sph_skein512 (&ctx_skein, input, 80);
	sph_skein512_close(&ctx_skein, (void*) hash);
	
	sph_jh512_init(&ctx_jh);
	sph_jh512 (&ctx_jh, (const void*) hash, 64);
	sph_jh512_close(&ctx_jh, (void*) hash);
	
	sph_keccak512_init(&ctx_keccak);
	sph_keccak512 (&ctx_keccak, (const void*) hash, 64);
	sph_keccak512_close(&ctx_keccak, (void*) hash);
	
	sph_bmw512_init(&ctx_bmw);
	sph_bmw512 (&ctx_bmw, (const void*) hash, 64);
	sph_bmw512_close(&ctx_bmw, (void*) hash);
	
	sph_luffa512_init(&ctx_luffa);
	sph_luffa512 (&ctx_luffa, (const void*) hash, 64);
	sph_luffa512_close (&ctx_luffa, (void*) hash);
	
	sph_echo512_init(&ctx_echo);
	sph_echo512 (&ctx_echo, (const void*) hash, 64);
	sph_echo512_close(&ctx_echo, (void*) hash);
	
	sph_simd512_init(&ctx_simd);
	sph_simd512 (&ctx_simd, (const void*) hash, 64);
	sph_simd512_close(&ctx_simd, (void*) hash);
	
	sph_shavite512_init(&ctx_shavite);
	sph_shavite512 (&ctx_shavite, (const void*) hash, 64);
	sph_shavite512_close(&ctx_shavite, (void*) hash);

	memcpy(state, hash, 32);
}

static bool init[MAX_GPUS] = { 0 };

extern "C" int scanhash_qubit(int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done)
{
	uint32_t _ALIGN(64) endiandata[20];
	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;
	const uint32_t first_nonce = pdata[19];
	uint32_t throughput =  cuda_default_throughput(thr_id, 1U << 19); // 256*256*8
	if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);

	if (opt_benchmark)
		ptarget[7] = 0x007f;

	if (!init[thr_id])
	{
		hipSetDevice(device_map[thr_id]);
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			// reduce cpu usage
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
			CUDA_LOG_ERROR();
		}
		gpulog(LOG_INFO, thr_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput), throughput);

		quark_skein512_cpu_init(thr_id, throughput);
		quark_jh512_cpu_init(thr_id, throughput);
		quark_keccak512_cpu_init(thr_id, throughput);
		quark_bmw512_cpu_init(thr_id, throughput);		
		x11_luffa512_cpu_init(thr_id, throughput);
		x11_echo512_cpu_init(thr_id, throughput);
		x11_simd512_cpu_init(thr_id, throughput);
		x11_shavite512_cpu_init(thr_id, throughput);

		CUDA_CALL_OR_RET_X(hipMalloc(&d_hash[thr_id], (size_t) 64 * throughput), 0);

		cuda_check_cpu_init(thr_id, throughput);

		init[thr_id] = true;
	}

	for (int k=0; k < 20; k++)
		be32enc(&endiandata[k], pdata[k]);

	skein512_cpu_setBlock_80((void*)endiandata);
	cuda_check_cpu_setTarget(ptarget);

	do {
		int order = 0;

		// Hash with CUDA

		skein512_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
		quark_jh512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);		
		quark_keccak512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		quark_bmw512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		x11_luffa512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		x11_echo512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);		
		x11_simd512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		x11_shavite512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);

		*hashes_done = pdata[19] - first_nonce + throughput;

		work->nonces[0] = cuda_check_hash(thr_id, throughput, pdata[19], d_hash[thr_id]);
		if (work->nonces[0] != UINT32_MAX)
		{
			const uint32_t Htarg = ptarget[7];
			uint32_t _ALIGN(64) vhash[8];
			be32enc(&endiandata[19], work->nonces[0]);
			qubithash(vhash, endiandata);

			if (vhash[7] <= Htarg && fulltest(vhash, ptarget)) {
				work->valid_nonces = 1;
				work_set_target_ratio(work, vhash);
				work->nonces[1] = cuda_check_hash_suppl(thr_id, throughput, pdata[19], d_hash[thr_id], 1);
				if (work->nonces[1] != 0) {
					be32enc(&endiandata[19], work->nonces[1]);
					qubithash(vhash, endiandata);
					bn_set_target_ratio(work, vhash, 1);
					work->valid_nonces++;
					pdata[19] = max(work->nonces[0], work->nonces[1]) + 1;
				} else {
					pdata[19] = work->nonces[0] + 1; // cursor
				}
				return work->valid_nonces;
			}
			else if (vhash[7] > Htarg) {
				gpu_increment_reject(thr_id);
				if (!opt_quiet)
				gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU!", work->nonces[0]);
				pdata[19] = work->nonces[0] + 1;
				continue;
			}
		}

		if ((uint64_t) throughput + pdata[19] >= max_nonce) {
			pdata[19] = max_nonce;
			break;
		}
		pdata[19] += throughput;

	} while (!work_restart[thr_id].restart);

	*hashes_done = pdata[19] - first_nonce;
	return 0;
}

// cleanup
extern "C" void free_qubit(int thr_id)
{
	if (!init[thr_id])
		return;

	hipDeviceSynchronize();

	hipFree(d_hash[thr_id]);

	x11_simd512_cpu_free(thr_id);

	cuda_check_cpu_free(thr_id);
	init[thr_id] = false;

	hipDeviceSynchronize();
}
